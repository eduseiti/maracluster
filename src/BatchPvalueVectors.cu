#include "hip/hip_runtime.h"
/******************************************************************************  
  Copyright 2015 Matthew The <matthew.the@scilifelab.se>
  Licensed under the Apache License, Version 2.0 (the "License");
  you may not use this file except in compliance with the License.
  You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

  Unless required by applicable law or agreed to in writing, software
  distributed under the License is distributed on an "AS IS" BASIS,
  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
  See the License for the specific language governing permissions and
  limitations under the License.
  
 ******************************************************************************/
 
#include "BatchPvalueVectors.cuh"

namespace maracluster {

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void polyval(double *polyfit, double *x) {
  // Horner's method
  double y = polyfit[POLYFIT_SIZE - 1];
  for (int i = POLYFIT_SIZE - 2; i >= 0; --i) {
    y = polyfit[i] + y*(*x);
  }
  if (y > 0.0) {
    *x = 0.0;
  } else {
    *x = y;
  }
}

__device__
void binaryMatchPeakBins(short *targetPeakBins, short *targetPeakScores, short *queryPeakBins, int *score) {
  size_t qIdx = 0;
  for (size_t i = 0; i < SCORING_PEAKS; ++i) {
    if (qIdx >= SCORING_PEAKS) break;
    while (targetPeakBins[i] > queryPeakBins[qIdx]) {
      if (++qIdx >= SCORING_PEAKS) break;
    }
    if (qIdx >= SCORING_PEAKS) break;
    if (targetPeakBins[i] == queryPeakBins[qIdx]) {
      ++qIdx;
    } else {
      *score += targetPeakScores[i];
    }
  }
}

__global__
void calculatePvals(short *peakBins, short *peakScores, double *polyfits, int *maxScores, int n, short *queryPeakBins, int m, double *pvals) {
  size_t targetIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  size_t queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  
  if (targetIdx < n && queryIdx < m) {
    size_t flattenedTargetIdx = targetIdx * SCORING_PEAKS;
    size_t flattenedQueryIdx = queryIdx * SCORING_PEAKS;
    
    int score = 0;
    binaryMatchPeakBins(&peakBins[flattenedTargetIdx], &peakScores[flattenedTargetIdx], &queryPeakBins[flattenedQueryIdx], &score);
    
    double relScore = static_cast<double>(score) / maxScores[targetIdx];
    polyval(&polyfits[targetIdx * POLYFIT_SIZE], &relScore);
    size_t pvalIdx = targetIdx * m + queryIdx;
    pvals[pvalIdx] = relScore;
  }
}

__host__
void runKernel(short *peakBins, short *peakScores, double *polyfits, int *maxScores, size_t N, short *queryPeakBins, size_t M, double *pvals) {
  short *peakBinsDevice, *peakScoresDevice, *queryPeakBinsDevice;
  double *polyfitsDevice, *pvalsDevice;
  int *maxScoresDevice;
  
  gpuErrchk( hipMalloc(&peakBinsDevice, N * SCORING_PEAKS * sizeof(short)) );
  gpuErrchk( hipMalloc(&peakScoresDevice, N * SCORING_PEAKS * sizeof(short)) );
  gpuErrchk( hipMalloc(&polyfitsDevice, N * POLYFIT_SIZE * sizeof(double)) );
  gpuErrchk( hipMalloc(&maxScoresDevice, N * sizeof(int)) );
  gpuErrchk( hipMalloc(&pvalsDevice, N * PVEC_MAX_BATCH_SIZE * sizeof(double)) );
  gpuErrchk( hipMalloc(&queryPeakBinsDevice, M * SCORING_PEAKS * sizeof(short)) );
  
  gpuErrchk( hipMemcpy(peakBinsDevice, peakBins, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(peakScoresDevice, peakScores, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(polyfitsDevice, polyfits, N * POLYFIT_SIZE * sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(maxScoresDevice, maxScores, N * sizeof(int), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(queryPeakBinsDevice, queryPeakBins, M * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid;
  dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;
  
  calculatePvals<<<dimGrid, dimBlock>>>(peakBinsDevice, peakScoresDevice, polyfitsDevice, maxScoresDevice, N, queryPeakBinsDevice, M, pvalsDevice);
  
  //gpuErrchk( hipDeviceSynchronize() );
  
  gpuErrchk( hipMemcpy(pvals, pvalsDevice, N * PVEC_MAX_BATCH_SIZE * sizeof(double), hipMemcpyDeviceToHost) );
  
  /*
  double sumPvals = 0;
  for (size_t i = 0; i < N; i++) {
    sumPvals += pvals[i*M];
  }
  std::cout << "Sum: " << sumPvals << std::endl;
  */
  
  // Free memory
  gpuErrchk( hipFree(peakBinsDevice) );
  gpuErrchk( hipFree(peakScoresDevice) );
  gpuErrchk( hipFree(polyfitsDevice) );
  gpuErrchk( hipFree(maxScoresDevice) );
  gpuErrchk( hipFree(pvalsDevice) );
  gpuErrchk( hipFree(queryPeakBinsDevice) );
}

} /* namespace maracluster */
