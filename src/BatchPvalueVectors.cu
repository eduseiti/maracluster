#include "hip/hip_runtime.h"
/******************************************************************************  
  Copyright 2015 Matthew The <matthew.the@scilifelab.se>
  Licensed under the Apache License, Version 2.0 (the "License");
  you may not use this file except in compliance with the License.
  You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

  Unless required by applicable law or agreed to in writing, software
  distributed under the License is distributed on an "AS IS" BASIS,
  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
  See the License for the specific language governing permissions and
  limitations under the License.
  
 ******************************************************************************/
 
#include "BatchPvalueVectors.cuh"

namespace maracluster {

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void polyval(double *polyfit, double *x) {
  // Horner's method
  double y = polyfit[POLYFIT_SIZE - 1];
  for (int i = POLYFIT_SIZE - 2; i >= 0; --i) {
    y = polyfit[i] + y*(*x);
  }
  if (y > 0.0) {
    *x = 0.0;
  } else {
    *x = y;
  }
}

__device__
void binaryMatchPeakBins(short *targetPeakBins, short *targetPeakScores, short *queryPeakBins, int *score) {
  size_t qIdx = 0;
  for (size_t i = 0; i < SCORING_PEAKS; ++i) {
    if (qIdx >= SCORING_PEAKS) break;
    while (targetPeakBins[i] > queryPeakBins[qIdx]) {
      if (++qIdx >= SCORING_PEAKS) break;
    }
    if (qIdx >= SCORING_PEAKS) break;
    if (targetPeakBins[i] == queryPeakBins[qIdx]) {
      ++qIdx;
    } else {
      *score += targetPeakScores[i];
    }
  }
}

__global__
void calculatePvals(short *peakBins, short *peakScores, double *polyfits, int *maxScores, int n, short *queryPeakBins, int m, double *pvals) {
  size_t targetIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  size_t queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  
  if (targetIdx < n && queryIdx < m) {
    size_t flattenedTargetIdx = targetIdx * SCORING_PEAKS;
    size_t flattenedQueryIdx = queryIdx * SCORING_PEAKS;
    
    int score = 0;
    binaryMatchPeakBins(&peakBins[flattenedTargetIdx], &peakScores[flattenedTargetIdx], &queryPeakBins[flattenedQueryIdx], &score);
    
    double relScore = static_cast<double>(score) / maxScores[targetIdx];
    polyval(&polyfits[targetIdx * POLYFIT_SIZE], &relScore);
    size_t pvalIdx = targetIdx * m + queryIdx;
    pvals[pvalIdx] = relScore;
  }
}

float totaltime = 0.0f;
hipStream_t streams[NUM_STREAMS];
short *peakBinsDevice[NUM_STREAMS];
short *peakScoresDevice[NUM_STREAMS];
short *queryPeakBinsDevice[NUM_STREAMS];
double *polyfitsDevice[NUM_STREAMS];
double *pvalsDevice[NUM_STREAMS];
int *maxScoresDevice[NUM_STREAMS];
  
void initStreams(double **pvalsHost) {
  for (int i = 0; i < NUM_STREAMS; ++i) { 
    gpuErrchk( hipStreamCreate(&streams[i]) );
    gpuErrchk( hipMalloc(&peakBinsDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
    gpuErrchk( hipMalloc(&peakScoresDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
    gpuErrchk( hipMalloc(&polyfitsDevice[i], PVEC_MAX_BATCH_SIZE * POLYFIT_SIZE * sizeof(double)) );
    gpuErrchk( hipMalloc(&maxScoresDevice[i], PVEC_MAX_BATCH_SIZE * sizeof(int)) );
    gpuErrchk( hipMalloc(&pvalsDevice[i], PVEC_MAX_BATCH_SIZE * PVEC_MAX_BATCH_SIZE * sizeof(double)) );
    gpuErrchk( hipHostMalloc(&pvalsHost[i], PVEC_MAX_BATCH_SIZE * PVEC_MAX_BATCH_SIZE * sizeof(double)) );
    gpuErrchk( hipMalloc(&queryPeakBinsDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
  }
}

void destroyStreams(double **pvalsHost) {
  for (int i = 0; i < NUM_STREAMS; ++i) { 
    gpuErrchk( hipStreamDestroy(streams[i]) );
    // Free memory
    gpuErrchk( hipFree(peakBinsDevice[i]) );
    gpuErrchk( hipFree(peakScoresDevice[i]) );
    gpuErrchk( hipFree(polyfitsDevice[i]) );
    gpuErrchk( hipFree(maxScoresDevice[i]) );
    gpuErrchk( hipFree(pvalsDevice[i]) );
    gpuErrchk( hipHostFree(pvalsHost[i]) );
    gpuErrchk( hipFree(queryPeakBinsDevice[i]) );
  }
}

void synchronizeStream(int streamIdx) {
  hipStreamSynchronize(streams[streamIdx]);
}

__host__
void runKernel(short *peakBins, short *peakScores, double *polyfits, int *maxScores, size_t N, short *queryPeakBins, size_t M, double** pvalsHost, int streamIdx) {
  hipSetDevice(0);
  
  hipStream_t *stream = &streams[streamIdx];
  /*
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  hipEventRecord(start);
  */
  gpuErrchk( hipMemcpy(peakBinsDevice[streamIdx], peakBins, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(peakScoresDevice[streamIdx], peakScores, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(polyfitsDevice[streamIdx], polyfits, N * POLYFIT_SIZE * sizeof(double), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(maxScoresDevice[streamIdx], maxScores, N * sizeof(int), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(queryPeakBinsDevice[streamIdx], queryPeakBins, M * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice) );
  
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid;
  dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;  
  
  calculatePvals<<<dimGrid, dimBlock, 0, *stream>>>(peakBinsDevice[streamIdx], peakScoresDevice[streamIdx], polyfitsDevice[streamIdx], maxScoresDevice[streamIdx], N, queryPeakBinsDevice[streamIdx], M, pvalsDevice[streamIdx]);
  
  
  //gpuErrchk( hipDeviceSynchronize() );
  
  gpuErrchk( hipMemcpyAsync(pvalsHost[streamIdx], pvalsDevice[streamIdx], N * PVEC_MAX_BATCH_SIZE * sizeof(double), hipMemcpyDeviceToHost, *stream) );
  /*
  double sumPvals = 0;
  for (size_t i = 0; i < N; i++) {
    sumPvals += pvals[i*M];
  }
  std::cout << "Sum: " << sumPvals << std::endl;
  */
  
  /*
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  totaltime += milliseconds;
  
  std::cout << "Elapsed time (ms): " << totaltime << std::endl;
  */
}

} /* namespace maracluster */
