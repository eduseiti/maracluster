#include "hip/hip_runtime.h"
/******************************************************************************  
  Copyright 2015 Matthew The <matthew.the@scilifelab.se>
  Licensed under the Apache License, Version 2.0 (the "License");
  you may not use this file except in compliance with the License.
  You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

  Unless required by applicable law or agreed to in writing, software
  distributed under the License is distributed on an "AS IS" BASIS,
  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
  See the License for the specific language governing permissions and
  limitations under the License.
  
 ******************************************************************************/
 
#include "BatchPvalueVectors.cuh"

namespace maracluster {

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__
void polyval(double *polyfit, double *x) {
  // Horner's method
  double y = polyfit[POLYFIT_SIZE - 1];
  for (int i = POLYFIT_SIZE - 2; i >= 0; --i) {
    y = polyfit[i] + y*(*x);
  }
  if (y > 0.0) {
    *x = 0.0;
  } else {
    *x = y;
  }
}

__device__
void binaryMatchPeakBinsOld(short *targetPeakBins, short *targetPeakScores, short *queryPeakBins, int *score) {
  size_t qIdx = 0;
  for (size_t i = 0; i < SCORING_PEAKS; ++i) {
    if (qIdx >= SCORING_PEAKS) break;
    while (targetPeakBins[i] > queryPeakBins[qIdx]) {
      if (++qIdx >= SCORING_PEAKS) break;
    }
    if (qIdx >= SCORING_PEAKS) break;
    if (targetPeakBins[i] == queryPeakBins[qIdx]) {
      ++qIdx;
      *score += targetPeakScores[i];
    }
  }
}

__device__
void binaryMatchPeakBins(short *targetPeakBins, short *targetPeakScores, short *queryPeakBins, int *score) {
  size_t tIdx = 0, qIdx = 0;
  short t = -1, q = -1;
  while (tIdx < SCORING_PEAKS && qIdx < SCORING_PEAKS) {
    t = targetPeakBins[tIdx];
    q = queryPeakBins[qIdx];
    if (t == q) {
      *score += targetPeakScores[tIdx];
      ++tIdx;
      ++qIdx;
    } else {
      tIdx += (t < q);
      qIdx += (q < t);
    }
  }
}

__global__
void calculatePvals(short *peakBins, short *peakScores, double *polyfits, int *maxScores, int n, short *queryPeakBins, int m, double *pvals) {
  __shared__ short peakBinsShared[BLOCK_SIZE][SCORING_PEAKS];
  __shared__ short peakScoresShared[BLOCK_SIZE][SCORING_PEAKS];
  __shared__ double polyfitsShared[BLOCK_SIZE][POLYFIT_SIZE];
  __shared__ int maxScoresShared[BLOCK_SIZE];
  __shared__ short queryPeakBinsShared[BLOCK_SIZE][SCORING_PEAKS];
  
  size_t targetIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  size_t queryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  
  size_t flattenedTargetIdx = targetIdx * SCORING_PEAKS;
  size_t flattenedQueryIdx = queryIdx * SCORING_PEAKS;
  
  size_t placeHolderQueryIdx = blockIdx.y * BLOCK_SIZE + threadIdx.x;
  size_t placeholderFlattenedQueryIdx = placeHolderQueryIdx * SCORING_PEAKS;
  
  if (threadIdx.y == 0 && targetIdx < n) {
    for (int i = 0; i < SCORING_PEAKS; ++i) {
      peakBinsShared[threadIdx.x][i] = peakBins[flattenedTargetIdx + i];
    }
  }
  if (threadIdx.y == 2 && targetIdx < n) {
    for (int i = 0; i < SCORING_PEAKS; ++i) {
      peakScoresShared[threadIdx.x][i] = peakScores[flattenedTargetIdx + i];
    }
  }
  if (threadIdx.y == 4 && targetIdx < n) {
    for (int i = 0; i < POLYFIT_SIZE; ++i) {
      polyfitsShared[threadIdx.x][i] = polyfits[targetIdx * POLYFIT_SIZE + i];
    }
    maxScoresShared[threadIdx.x] = maxScores[targetIdx];
  }
  if (threadIdx.y == 6 && placeHolderQueryIdx < m) {
    for (int i = 0; i < SCORING_PEAKS; ++i) {
      queryPeakBinsShared[threadIdx.x][i] = queryPeakBins[placeholderFlattenedQueryIdx + i];
    }
  }
  __syncthreads();
  if (targetIdx < n && queryIdx < m) {  
    int score = 0;
    binaryMatchPeakBins(&peakBinsShared[threadIdx.x][0], &peakScoresShared[threadIdx.x][0], &queryPeakBinsShared[threadIdx.y][0], &score);
    //binaryMatchPeakBins(&peakBinsShared[threadIdx.x][0], &peakScores[flattenedTargetIdx], &queryPeakBinsShared[threadIdx.y][0], &score);
    //binaryMatchPeakBins(&peakBinsShared[threadIdx.x][0], &peakScores[flattenedTargetIdx], &queryPeakBins[flattenedQueryIdx], &score);
    //binaryMatchPeakBins(&peakBins[flattenedTargetIdx], &peakScores[flattenedTargetIdx], &queryPeakBins[flattenedQueryIdx], &score);
    
    double relScore = static_cast<double>(maxScoresShared[threadIdx.x] - score) / maxScoresShared[threadIdx.x];
    //double relScore = static_cast<double>(maxScores[targetIdx] - score) / maxScores[targetIdx];
    //double relScore = static_cast<double>(score) / maxScores[targetIdx];
    polyval(&polyfitsShared[threadIdx.x][0], &relScore);
    //polyval(&polyfits[targetIdx * POLYFIT_SIZE], &relScore);
    size_t pvalIdx = targetIdx * PVEC_MAX_BATCH_SIZE + queryIdx;
    pvals[pvalIdx] = static_cast<double>(relScore);
  }
}

float totaltime = 0.0f;
hipStream_t streams[NUM_STREAMS];
short *peakBinsDevice[NUM_STREAMS];
short *peakScoresDevice[NUM_STREAMS];
short *queryPeakBinsDevice[NUM_STREAMS];
double *polyfitsDevice[NUM_STREAMS];
double *pvalsDevice[NUM_STREAMS];
int *maxScoresDevice[NUM_STREAMS];
  
void initStreams(double **pvalsHost) {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  std::cerr << "#GPUs: " << nDevices << std::endl;
  for (int i = 0; i < NUM_STREAMS; ++i) { 
    gpuErrchk( hipSetDevice(i % NUM_DEVICES) );
    gpuErrchk( hipStreamCreate(&streams[i]) );
    gpuErrchk( hipMalloc(&peakBinsDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
    gpuErrchk( hipMalloc(&peakScoresDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
    gpuErrchk( hipMalloc(&polyfitsDevice[i], PVEC_MAX_BATCH_SIZE * POLYFIT_SIZE * sizeof(double)) );
    gpuErrchk( hipMalloc(&maxScoresDevice[i], PVEC_MAX_BATCH_SIZE * sizeof(int)) );
    gpuErrchk( hipMalloc(&pvalsDevice[i], PVEC_MAX_BATCH_SIZE * PVEC_MAX_BATCH_SIZE * sizeof(double)) );
    gpuErrchk( hipHostMalloc(&pvalsHost[i], PVEC_MAX_BATCH_SIZE * PVEC_MAX_BATCH_SIZE * sizeof(double)) );
    gpuErrchk( hipMalloc(&queryPeakBinsDevice[i], PVEC_MAX_BATCH_SIZE * SCORING_PEAKS * sizeof(short)) );
  }
}

void destroyStreams(double **pvalsHost) {
  for (int i = 0; i < NUM_STREAMS; ++i) {
    gpuErrchk( hipSetDevice(i % NUM_DEVICES) );
    gpuErrchk( hipStreamDestroy(streams[i]) );
    // Free memory
    gpuErrchk( hipFree(peakBinsDevice[i]) );
    gpuErrchk( hipFree(peakScoresDevice[i]) );
    gpuErrchk( hipFree(polyfitsDevice[i]) );
    gpuErrchk( hipFree(maxScoresDevice[i]) );
    gpuErrchk( hipFree(pvalsDevice[i]) );
    gpuErrchk( hipHostFree(pvalsHost[i]) );
    gpuErrchk( hipFree(queryPeakBinsDevice[i]) );
  }
}

void synchronizeStream(int streamIdx) {
  hipStreamSynchronize(streams[streamIdx]);
}

__host__
void runKernel(short *peakBins, short *peakScores, double *polyfits, int *maxScores, size_t N, short *queryPeakBins, size_t M, double** pvalsHost, int streamIdx) {
  gpuErrchk( hipSetDevice(streamIdx % NUM_DEVICES) );
  
  hipStream_t *stream = &streams[streamIdx];
  /*
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  */
  
  /* These hipMemcpyAsync probably do not work exactly as expected, as the host memory is not pinned. 
     However, it needs to be on the same stream as the kernel to ensure the copies are finished before the kernel executes, which occurs when using hipMemcpy with data transfers below 64KB */
  gpuErrchk( hipMemcpyAsync(peakBinsDevice[streamIdx], peakBins, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice, *stream) );
  gpuErrchk( hipMemcpyAsync(peakScoresDevice[streamIdx], peakScores, N * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice, *stream) );
  gpuErrchk( hipMemcpyAsync(polyfitsDevice[streamIdx], polyfits, N * POLYFIT_SIZE * sizeof(double), hipMemcpyHostToDevice, *stream) );
  gpuErrchk( hipMemcpyAsync(maxScoresDevice[streamIdx], maxScores, N * sizeof(int), hipMemcpyHostToDevice, *stream) );
  gpuErrchk( hipMemcpyAsync(queryPeakBinsDevice[streamIdx], queryPeakBins, M * SCORING_PEAKS * sizeof(short), hipMemcpyHostToDevice, *stream) );
  
  //gpuErrchk( hipDeviceSynchronize() );
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid;
  dimGrid.x = (N + dimBlock.x - 1) / dimBlock.x;
  dimGrid.y = (M + dimBlock.y - 1) / dimBlock.y;  
  
  
  calculatePvals<<<dimGrid, dimBlock, 0, *stream>>>(peakBinsDevice[streamIdx], peakScoresDevice[streamIdx], polyfitsDevice[streamIdx], maxScoresDevice[streamIdx], N, queryPeakBinsDevice[streamIdx], M, pvalsDevice[streamIdx]); 
  
  
  //gpuErrchk( hipDeviceSynchronize() );
  //hipEventRecord(start);
  gpuErrchk( hipMemcpyAsync(pvalsHost[streamIdx], pvalsDevice[streamIdx], N * PVEC_MAX_BATCH_SIZE * sizeof(double), hipMemcpyDeviceToHost, *stream) );
  /*
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  totaltime += milliseconds;
  
  std::cout << "Elapsed time (ms): " << totaltime << std::endl;
  */
  /*
  double sumPvals = 0;
  for (size_t i = 0; i < N; i++) {
    sumPvals += pvals[i*M];
  }
  std::cout << "Sum: " << sumPvals << std::endl;
  */
  
  /*
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  
  totaltime += milliseconds;
  
  std::cout << "Elapsed time (ms): " << totaltime << std::endl;
  */
}

} /* namespace maracluster */
